#include "hip/hip_runtime.h"
#include "types.h"
#include "common.h"
#include "imageproc.h"
#include "counter.h"
#include "cudawrapper.hpp"

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define CMIN(a,b) (((a)<(b))?(a):(b))
#define CMAX(a,b) (((a)>(b))?(a):(b))

__global__ void gaussianBlurCuda(uchar *src, uchar *dest, int w, int h, int r)
{
	const int col = blockDim.x * blockIdx.x + threadIdx.x;
	const int row = blockDim.y * blockIdx.y + threadIdx.y;

	if (col > w || row > h)
		return;

	auto rs = std::ceil(r * 2.57);

	for(int i = 0; i < h; i++)
	{
		for(int j = 0; j < w; j++)
		{
			double val = 0;
			double wsum = 0;
			for(int iy = i - rs; iy < i + rs + 1; iy++)
			{
				for(int ix = j - rs; ix < j + rs + 1; ix++)
				{
					int x = CMIN(w - 1 + col, CMAX(0, ix));
					int y = CMIN(h - 1 + row, CMAX(0, iy));
					double dsq = (ix - j) * (ix - j) + (iy - i) * (iy - i);
					double wght = std::exp(-dsq / (2 * r * r )) / (M_PI * 2 * r * r);
					val += src[y * w + x] * wght;
					wsum += wght;
				}
			}

			dest[i * w + j] = std::round(val / wsum);
		}
	}
}

void processCuda(const string &fileName)
{
	std::cout << "Start processing image with cuda .." << std::endl;
	std::cout << "File: " << fileName << std::endl;

	if (!isFileExists(fileName))
	{
		std::cout << "Can't find file: " << fileName << std::endl;
		return;
	}

	cv::Mat image = cv::imread(fileName);
	if (image.empty() || image.size().empty())
	{
		std::cout << "Can't load image" << std::endl;
		return;
	}

	Counter counter;
	ImageProc imageProc(fileName);
	imageProc.processGrayscale(image);

	size_t size = image.total();
	ByteArray dest(size, 0);
	ByteArray src(image.data, image.data + size);

	hipError_t error = hipSuccess;
	uchar *devDest = NULL;
	uchar *devSrc = NULL;

	int deviceCount = 0;
	error = hipGetDeviceCount(&deviceCount);
	if (error != hipSuccess)
	{
		std::cout << "hipGetDeviceCount error " << error << std::endl;
		return;
	}

	if (deviceCount == 0)
	{
		std::cout << "There is no device!" << std::endl;
		return;
	}

	error = hipMalloc(&devDest, size);
	if (error != hipSuccess)
	{
		std::cout << "hipMalloc error " << error << std::endl;
		return;
	}

	error = hipMalloc(&devSrc, size);
	if (error != hipSuccess)
	{
		std::cout << "hipMalloc error " << error << std::endl;
		return;
	}

	uchar *srcData = src.data();
	error = hipMemcpy(devSrc, srcData, size, hipMemcpyHostToDevice);
	if (error != hipSuccess)
	{
		std::cout << "hipMemcpy error " << error << std::endl;
		return;
	}

	int threadsPerBlock = 256;
	int blocksPerGrid = (image.total() + threadsPerBlock - 1) / threadsPerBlock;
	gaussianBlurCuda<<<blocksPerGrid, threadsPerBlock>>>(devSrc, devDest, image.cols, image.rows, 15);

	uchar *destData = dest.data();
	hipMemcpy(destData, devDest, dest.size(), hipMemcpyDeviceToHost);
	cv::Mat newImage(image.rows, image.cols, CV_8U, destData);

	counter.show();
	string outFile = fileName + "-new.jpg";
	cv::imwrite(outFile, newImage);
	std::cout << "All done, written file: " << outFile << std::endl;

	hipFree(devSrc);
	hipFree(devDest);
}
